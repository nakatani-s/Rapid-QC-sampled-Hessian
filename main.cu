#include "hip/hip_runtime.h"
/*
  Least Squares Method: LSM 最小二乗法
  QuadraticHyperPlane:  QHP ２次超平面
*/ 

#include<iostream>
#include <stdio.h>
#include <fstream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <errno.h>
#include <string.h>
#include <unistd.h>
#include <iomanip>

#include "include/params.cuh"
#include "include/init.cuh"
#include "include/DataStructure.cuh"
#include "include/MCMPC.cuh"
#include "include/LSM_QuadHyperPlane.cuh"
#include "include/Matrix.cuh"
#include "include/costFunction.cuh"

#define CHECK(call)                                                  \
{                                                                    \
    const hipError_t error = call;                                  \
    if (error != hipSuccess)                                        \
    {                                                                \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                \
        printf("code:%d, reason: %s\n", error,                       \
                hipGetErrorString(error));                          \
        exit(1);                                                     \
    }                                                                \
}
#define CHECK_CUBLAS(call,str)                                                        \
{                                                                                     \
    if ( call != HIPBLAS_STATUS_SUCCESS)                                               \
    {                                                                                 \
        printf("CUBLAS Error: %s : %s %d\n", str, __FILE__, __LINE__);                \
        exit(1);                                                                      \
    }                                                                                 \
}

#define CHECK_CUSOLVER(call,str)                                                      \
{                                                                                     \
    if ( call != HIPSOLVER_STATUS_SUCCESS)                                             \
    {                                                                                 \
        printf("CUBLAS Error: %s : %s %d\n", str, __FILE__, __LINE__);                \
        exit(1);                                                                      \
    }                                                                                 \
}





int main(int argc, char **argv)
{
    /* 行列演算ライブラリを使用するために宣言 */
    hipsolverHandle_t cusolverH = NULL;
    // hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; 
    CHECK_CUSOLVER( hipsolverDnCreate(&cusolverH),"Failed to Create cusolver handle");

    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    /* データ書き込み用ファイルの定義 */
    FILE *fp;
    time_t timeValue;
    struct tm *timeObject;
    time( &timeValue );
    timeObject = localtime( &timeValue );
    char filename1[35];
    sprintf(filename1,"data_system_%d%d_%d%d.txt",timeObject->tm_mon + 1, timeObject->tm_mday, timeObject->tm_hour,timeObject->tm_min);
    fp = fopen(filename1,"w");


    /* ホスト・デバイスで使用するベクトルの宣言 */
    float hostParams[DIM_OF_PARAMETERS], hostState[DIM_OF_STATES], hostConstraint[NUM_OF_CONSTRAINTS], hostWeightMatrix[DIM_OF_WEIGHT_MATRIX];
    float *deviceParams, *deviceState, *deviceConstraint, *deviceWeightMatrix;
    initialize_host_vector(hostParams, hostState, hostConstraint, hostWeightMatrix);
    hipMalloc(&deviceParams, sizeof(float) * DIM_OF_PARAMETERS);
    hipMalloc(&deviceState, sizeof(float) * DIM_OF_STATES);
    hipMalloc(&deviceConstraint, sizeof(float) * NUM_OF_CONSTRAINTS);
    hipMalloc(&deviceWeightMatrix, sizeof(float) * DIM_OF_WEIGHT_MATRIX);
    hipMemcpy(deviceParams, hostParams, sizeof(float) * DIM_OF_PARAMETERS, hipMemcpyHostToDevice);
    hipMemcpy(deviceState, hostState, sizeof(float) * DIM_OF_STATES, hipMemcpyHostToDevice);
    hipMemcpy(deviceConstraint, hostConstraint, sizeof(float) * NUM_OF_CONSTRAINTS, hipMemcpyHostToDevice);
    hipMemcpy(deviceWeightMatrix, hostWeightMatrix, sizeof(float)* DIM_OF_WEIGHT_MATRIX, hipMemcpyHostToDevice);

    /* GPUの設定用パラメータ */
    unsigned int numBlocks, randomBlocks, randomNums, Blocks, dimHessian, numUnknownParamQHP, numUnknownParamHessian;
    unsigned int paramsSizeQuadHyperPlane;
    randomNums = NUM_OF_SAMPLES * (DIM_OF_INPUT + 1) * HORIZON;
    randomBlocks = countBlocks(randomNums, THREAD_PER_BLOCKS);
    numBlocks = countBlocks(NUM_OF_SAMPLES, THREAD_PER_BLOCKS);
    Blocks = numBlocks;
    dimHessian = HORIZON * HORIZON;
    // numUnknownParamQHP = count_QHP_Parameters( HORIZON );
    numUnknownParamQHP = sizeOfParaboloidElements;
    numUnknownParamHessian = numUnknownParamQHP - (HORIZON + 1);
    paramsSizeQuadHyperPlane = numUnknownParamQHP; //ホライズンの大きさに併せて、局所サンプルのサイズを決定
    paramsSizeQuadHyperPlane = paramsSizeQuadHyperPlane + addTermForLSM;
    dim3 block(2,2);
    dim3 grid((numUnknownParamQHP + block.x - 1)/ block.x, (numUnknownParamQHP + block.y -1) / block.y);
    printf("#NumBlocks = %d\n", numBlocks);
    printf("#NumBlocks = %d\n", numUnknownParamQHP);

    /* GPUで乱数生成するために使用する乱数の種 */
    hiprandState *deviceRandomSeed;
    hipMalloc((void **)&deviceRandomSeed, randomNums * sizeof(hiprandState));
    setup_kernel<<<NUM_OF_SAMPLES, (DIM_OF_INPUT + 1) * HORIZON>>>(deviceRandomSeed, rand());
    hipDeviceSynchronize();

    /* sort用入力格納構造体の宣言と初期化 */
    InputVector *deviceInputSeq, *hostInputSeq, *deviceEliteInputSeq;
    hostInputSeq = (InputVector*)malloc(sizeof(InputVector) * NUM_OF_ELITES);
    hipMalloc(&deviceEliteInputSeq, sizeof(InputVector) * NUM_OF_ELITES);
    hipMalloc(&deviceInputSeq, sizeof(InputVector) * NUM_OF_SAMPLES);
    // init_Input_vector<<<NUM_OF_SAMPLES, 1>>>(deviceInputSeq, 0.0f);

    /* 2次超平面のパラメータ行列/ベクトル　（←最適値計算用に準備） */
    float *Hessian, *HessElements, *transGmatrix, *Hvector, *invGmHessSsymm /*, *Grad*/;
    hipMalloc(&Hessian, sizeof(float) * dimHessian );
    hipMalloc(&transGmatrix, sizeof(float) * dimHessian); /* Ans = -2 * G^T * Hessian * Hvector の　G^T  */
    hipMalloc(&Hvector, sizeof(float) * HORIZON ); /* Ans = -2 * G^T * Hessian * Hvector の　Hvector */
    hipMalloc(&invGmHessSsymm, sizeof(float) * dimHessian);
    // hipMalloc(&HessElements, sizeof(float) * numUnknownParamHessian);
    hipMalloc(&HessElements, sizeof(float) * numUnknownParamQHP );
    // hipMalloc(&Grad, sizeof(float) * HORIZON);
    /* 最小二乗法で2次超平面を求める際に使用 */
    float *Gmatrix, *invGmatrix, *Rvector, *ansRvector;
    CHECK(hipMalloc(&Rvector, sizeof(float) * numUnknownParamQHP));
    CHECK(hipMalloc(&ansRvector, sizeof(float) * numUnknownParamQHP));
    CHECK(hipMalloc(&Gmatrix, sizeof(float) * numUnknownParamQHP * numUnknownParamQHP));
    CHECK(hipMalloc(&invGmatrix, sizeof(float) * numUnknownParamQHP * numUnknownParamQHP) ); //elementsSize_QuadHyperPlaneMatrix = paramsSize_QuadHyperPlane * paramsSize_QuadHyperPlane
    //assert(hipSuccess == cudaStat2);
    QuadHyperPlane *deviceQuadHyPl;
    hipMalloc(&deviceQuadHyPl, sizeof(QuadHyperPlane) * paramsSizeQuadHyperPlane); //当面はブロック数分リサンプル　( HORIZON < Blocks < GPUコア数 で設計)
    unsigned int qhpBlocks;
    // qhpBlocks = countBlocks(numUnknownParamQHP, THREAD_PER_BLOCKS);
    qhpBlocks = countBlocks(paramsSizeQuadHyperPlane, THREAD_PER_BLOCKS);
    printf("#qhpblocks = %d\n", qhpBlocks);
    // float *KVALUE_MATRIX, *HESSIAN_MATRIX;
    // KVALUE_MATRIX = (float *)malloc(sizeof(float)*numUnknownParamQHP * numUnknownParamQHP);
    // HESSIAN_MATRIX = (float *)malloc(sizeof(float)*dimHessian);
    //KVALUE_MATRIX = (float *)malloc(sizeof(float)*dimHessian);
    // 行列演算ライブラリ使用用に定義
    const int m_RMatrix = numUnknownParamQHP;
    printf("#NumBlocks = %d\n", m_RMatrix);
    // const int lda_RMatrix = m_RMatrix;
    int work_size, work_size_season2;
    float *work_space, *work_space_season2;
    int *devInfo;
    int *devInfo_season2;
    hipMalloc ((void**)&devInfo_season2, sizeof(int));
    hipblasHandle_t handle_cublas = 0;
    hipblasCreate(&handle_cublas);
    float alpha;
    float beta;
    alpha = 1.0f;
    beta = 0.0f;
    hipMalloc ((void**)&devInfo, sizeof(int));


    /* thrust使用のためのホスト/デバイスベクトル */
    thrust::host_vector<int> indices_host_vec( NUM_OF_SAMPLES );
    thrust::device_vector<int> indices_device_vec = indices_host_vec;
    thrust::host_vector<float> sort_key_host_vec( NUM_OF_SAMPLES );
    thrust::device_vector<float> sort_key_device_vec = sort_key_host_vec; 
    
    /* 推定入力のプロット・データ転送用 */
    float *hostData, *deviceData;
    hostData = (float *)malloc(sizeof(float) * HORIZON);
    hipMalloc(&deviceData, sizeof(float) * HORIZON);
    for(int i = 0; i < HORIZON; i++){
        hostData[i] = 0.0f;
    }
    hipMemcpy(deviceData, hostData, sizeof(float) * HORIZON, hipMemcpyHostToDevice);
    
    // float variance;
    /* 制御ループ */
    float est_input = 0.0f;
    float MCMPC_U, Proposed_U;
    float costFromMCMPC, costFromQHPMethod;
    costFromMCMPC = 0.0f;
    costFromQHPMethod = 0.0f;
    float vars;

    int counter;
    float process_gpu_time, procedure_all_time;
    clock_t start_t, stop_t;
    hipEvent_t start, stop;

    for(int t = 0; t < TIME; t++){
        shift_Input_vec( hostData );
        hipMemcpy(deviceData, hostData, sizeof(float) * HORIZON, hipMemcpyHostToDevice);
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        start_t = clock();
        for(int repeat = 0; repeat < ITERATIONS; repeat++){
            /* サンプルベースニュートンメソッドの初期値を決定するMCMPC */
            vars = powf(0.95,repeat) * variance; 
            MCMPC_Crat_and_SinglePole<<<numBlocks, THREAD_PER_BLOCKS>>>(deviceState, deviceRandomSeed, deviceData, deviceInputSeq, vars, deviceParams, deviceConstraint, deviceWeightMatrix,
                thrust::raw_pointer_cast( sort_key_device_vec.data() ));
            /*MCMPC_Simple_NonLinear_Example<<<numBlocks, THREAD_PER_BLOCKS>>>(deviceState, deviceRandomSeed, deviceData, deviceInputSeq, variance, deviceParams, deviceConstraint, deviceWeightMatrix,
                thrust::raw_pointer_cast( sort_key_device_vec.data() ));*/
            hipDeviceSynchronize();
            thrust::sequence(indices_device_vec.begin(), indices_device_vec.end());
            thrust::sort_by_key(sort_key_device_vec.begin(), sort_key_device_vec.end(), indices_device_vec.begin());
            
            /* エリートサンプル分の入力・コスト値をコールバックする関数 */ 
            callback_elite_sample<<<NUM_OF_ELITES, 1>>>(deviceEliteInputSeq, deviceInputSeq, thrust::raw_pointer_cast(indices_device_vec.data()));
            hipDeviceSynchronize();
            hipMemcpy(hostInputSeq, deviceEliteInputSeq, sizeof(InputVector) * NUM_OF_ELITES, hipMemcpyDeviceToHost);
            weighted_mean(hostInputSeq, NUM_OF_ELITES, hostData);
            MCMPC_U = hostData[0];
            
            CHECK(hipMemcpy(deviceData, hostData, sizeof(float) * HORIZON, hipMemcpyHostToDevice));
            
            costFromMCMPC = calc_Cost_Cart_and_SinglePole(hostData, hostState, hostParams, hostConstraint, hostWeightMatrix);
            printf("%dth MCMPC estimation ended\n", t*repeat);
            

            if(repeat == ITERATIONS - 1){
                /* 推定値近傍をサンプル・評価する関数 */
                MCMPC_Crat_and_SinglePole<<<numBlocks, THREAD_PER_BLOCKS>>>(deviceState, deviceRandomSeed, deviceData, deviceInputSeq, neighborVar, deviceParams, deviceConstraint, deviceWeightMatrix,
                    thrust::raw_pointer_cast( sort_key_device_vec.data() ));
                /*MCMPC_Simple_NonLinear_Example<<<numBlocks, THREAD_PER_BLOCKS>>>(deviceState, deviceRandomSeed, deviceData, deviceInputSeq, neighborVar, deviceParams, deviceConstraint, deviceWeightMatrix,
                    thrust::raw_pointer_cast( sort_key_device_vec.data() ));*/
                hipDeviceSynchronize();
                thrust::sequence(indices_device_vec.begin(), indices_device_vec.end());
                thrust::sort_by_key(sort_key_device_vec.begin(), sort_key_device_vec.end(), indices_device_vec.begin());
                /*device_QuadHyPlに，最小二乗法の左辺(column)と右辺の行列(テンソル積)計算用のベクトル(tensor)を格納する*/
                printf("hoge here l 185\n");
                LSM_QHP_make_tensor_vector<<<qhpBlocks, THREAD_PER_BLOCKS>>>(deviceQuadHyPl, deviceInputSeq, thrust::raw_pointer_cast( indices_device_vec.data() ));
                printf("hoge here l 187\n");
                hipDeviceSynchronize();
            /* Gmatrix に正規行列（最小二乗法で使用する逆行列の逆行列）*/ 
                if(numUnknownParamQHP > 1024){
                    LSM_QHP_make_regular_matrix_over_ThreadPerBlockLimit<<<grid,block>>>(Gmatrix, deviceQuadHyPl, paramsSizeQuadHyperPlane, numUnknownParamQHP);
                }else{
                    LSM_QHP_make_regular_matrix<<<numUnknownParamQHP,numUnknownParamQHP>>>(Gmatrix, deviceQuadHyPl, paramsSizeQuadHyperPlane);
                }
                hipDeviceSynchronize();
                printf("hoge here l 193\n");

                // 最小二乗法の結果（ヘシアンの要素＋勾配＋定数）
                LSM_QHP_make_regular_vector<<<numUnknownParamQHP,1>>>(Rvector, deviceQuadHyPl, paramsSizeQuadHyperPlane);
                hipDeviceSynchronize();

            /* Gmatrixの逆行列を計算 */
                CHECK_CUSOLVER( hipsolverDnSpotrf_bufferSize(cusolverH, uplo, m_RMatrix, Gmatrix, m_RMatrix, &work_size), "Failed to get bufferSize");
                CHECK( hipMalloc((void**)&work_space, sizeof(float)*work_size));
	            //hipGetErrorString(cudaStat1);
                //assert(hipSuccess == cudaStat1);
                CHECK_CUSOLVER(hipsolverDnSpotrf(cusolverH, uplo, m_RMatrix, Gmatrix, m_RMatrix , work_space, work_size, devInfo), "Failed to inverse operation for G");
                
                // 逆行列を取得するための単位行列の生成
                if(numUnknownParamQHP > 1024){
                    setup_Identity_Matrix_overMaxThread<<<grid, block>>>(invGmatrix, numUnknownParamQHP); 
                }else{
                    setup_Identity_Matrix<<<numUnknownParamQHP, numUnknownParamQHP>>>(invGmatrix); // invGmatrixを単位行列に変換
                }
                hipDeviceSynchronize();
                CHECK_CUSOLVER( hipsolverDnSpotrs(cusolverH, uplo, m_RMatrix, m_RMatrix , Gmatrix, m_RMatrix, invGmatrix, m_RMatrix, devInfo), "Failed to get inverse Matrix G");

                //LSM_QHP_get_reslt_all_elements<<<numUnknownParamQHP,1>>>(HessElements, invGmatrix, Rvector);
                /* 最小二乗法の行列演算　ansRvector = invGmatrix * Rvector を計算 */ 
                CHECK_CUBLAS( hipblasSgemv(handle_cublas, HIPBLAS_OP_N, m_RMatrix, m_RMatrix, &alpha, invGmatrix, m_RMatrix, Rvector, 1, &beta, ansRvector , 1), "Failed to get Estimate Input Sequences");

                //assert(  cublas_status == HIPBLAS_STATUS_SUCCESS );
                LSM_QHP_get_reslt_all_elements<<<numUnknownParamHessian,1>>>(HessElements, ansRvector); //numUnknownParamHessian これが大きすぎる?
                hipDeviceSynchronize();
                LSM_QHP_get_Hessian_Result<<<HORIZON, HORIZON>>>( Hessian, HessElements);
                hipDeviceSynchronize();
                LSM_QHP_transpose<<<HORIZON, HORIZON>>>(transGmatrix, Hessian);
                hipDeviceSynchronize();
                LSM_QHP_make_symmetric<<<HORIZON, HORIZON>>>(transGmatrix, Hessian);
                // hipMemcpy(HESSIAN_MATRIX, transGmatrix, sizeof(float) * dimHessian, hipMemcpyDeviceToHost);
                // printMatrix(HORIZON,HORIZON,HESSIAN_MATRIX, HORIZON, "HESSIAN");
                // ヘッシアンの計算まで終了
                //LSM_Hessian_To_Positive_Symmetric<<<HORIZON, HORIZON>>>(transGmatrix);

                /* -2*Hessian * b^T の b^Tベクトルを作成 (Hvector　←　b^T) */
                LSM_QHP_make_bVector<<<HORIZON, 1>>>(Hvector, ansRvector, numUnknownParamHessian);

                multiply_matrix<<<HORIZON, HORIZON>>>(Hessian, 2.0f, transGmatrix);
                CHECK_CUSOLVER(hipsolverDnSpotrf_bufferSize(cusolverH, uplo, HORIZON, Hessian, HORIZON, &work_size_season2),"Failed to get bufferSize of Hessian");
                CHECK( hipMalloc((void**)&work_space_season2, sizeof(float)*work_size_season2) );
  
                CHECK_CUSOLVER(hipsolverDnSpotrf(cusolverH, uplo, HORIZON, Hessian, HORIZON, work_space_season2, work_size_season2, devInfo_season2), "Failed to inverse operation");
            
                setup_Identity_Matrix<<<HORIZON, HORIZON>>>(invGmHessSsymm);
                hipDeviceSynchronize();
                CHECK_CUSOLVER(hipsolverDnSpotrs(cusolverH, uplo, HORIZON, HORIZON, Hessian, HORIZON, invGmHessSsymm, HORIZON, devInfo_season2), "Failed to get inverse Matrix of H");
                // hipMemcpy(HESSIAN_MATRIX, invGmHessSsymm, sizeof(float) * dimHessian, hipMemcpyDeviceToHost);
                multiply_matrix<<<HORIZON, HORIZON>>>(transGmatrix, -1.0f, invGmHessSsymm);

                copy_inputSequences<<<numBlocks, THREAD_PER_BLOCKS>>>(deviceInputSeq, deviceData);
                CHECK_CUBLAS(hipblasSgemv(handle_cublas, HIPBLAS_OP_N, HORIZON, HORIZON, &alpha, transGmatrix, HORIZON, Hvector, 1, &beta,  deviceData, 1),"Failed to get Result");
                //cublas_status = hipblasSgemv(handle_cublas, HIPBLAS_OP_N, HORIZON, HORIZON, &alpha, invGmHessSsymm, HORIZON, Hvector, 1, &beta,  deviceData, 1);
                hipMemcpy(hostData, deviceData, sizeof(float) * HORIZON, hipMemcpyDeviceToHost);
                //costFromQHPMethod = calc_Cost_Simple_NonLinear_Example(hostData, hostState,  hostParams, hostWeightMatrix);
                costFromQHPMethod = calc_Cost_Cart_and_SinglePole(hostData, hostState, hostParams, hostConstraint, hostWeightMatrix);
                Proposed_U = hostData[0];
            }

        }
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&process_gpu_time, start, stop);
        stop_t = clock();
        procedure_all_time = stop_t - start_t;
        // 推定入力列の先頭をコピー
        if(costFromMCMPC < costFromQHPMethod || isnan(costFromQHPMethod)){
	        est_input = MCMPC_U;
            counter = 0;
	    }else{
            est_input = Proposed_U;
            counter = 1;
        }
        Runge_kutta_45_for_Secondary_system(hostState, est_input, hostParams, interval);
        /*float hostDiffState[DIM_OF_STATES] = { };
        calc_nonLinear_example(hostState, est_input, hostParams, hostDiffState);
        for(int k = 0; k < DIM_OF_STATES; k++){
            hostState[k] = hostState[k] + (interval * hostDiffState[k]);
        }*/
        hipMemcpy(deviceState, hostState, sizeof(float) * DIM_OF_STATES, hipMemcpyHostToDevice);
        fprintf(fp,"%f %f %f %f %f %f %f %f %f %f %f %f %f %d\n", interval * t, est_input, MCMPC_U, Proposed_U, hostState[0], hostState[1], hostState[2], hostState[3], costFromMCMPC, costFromQHPMethod, costFromMCMPC - costFromQHPMethod, process_gpu_time/1000,procedure_all_time / CLOCKS_PER_SEC, counter);
        printf("u == %f MCMPC == %f  Proposed == %f  MCMPC - Proposed == %f\n", est_input,  costFromMCMPC, costFromQHPMethod, costFromMCMPC - costFromQHPMethod);
    }

    if(cusolverH) hipsolverDnDestroy(cusolverH);
    if(handle_cublas) hipblasDestroy(handle_cublas);
    fclose(fp);
    hipDeviceReset();
    printf("%s\n", hipGetErrorString(hipGetLastError()));
    return 0;
}
